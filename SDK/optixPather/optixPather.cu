#include "hip/hip_runtime.h"
//
// Copyright (c) 2023, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
#include <optix.h>

#include "optixPather.h"
#include "random.h"

#include <sutil/vec_math.h>
#include <cuda/helpers.h>

extern "C" {
__constant__ Params params;
}

//------------------------------------------------------------------------------
//
// Orthonormal basis helper
//
//------------------------------------------------------------------------------


struct Onb
{
  __forceinline__ __device__ Onb(const float3& normal)
  {
	m_normal = normal;

	if( fabs(m_normal.x) > fabs(m_normal.z) )
	{
	  m_binormal.x = -m_normal.y;
	  m_binormal.y =  m_normal.x;
	  m_binormal.z =  0;
	}
	else
	{
	  m_binormal.x =  0;
	  m_binormal.y = -m_normal.z;
	  m_binormal.z =  m_normal.y;
	}

	m_binormal = normalize(m_binormal);
	m_tangent = cross( m_binormal, m_normal );
  }

  __forceinline__ __device__ void inverse_transform(float3& p) const
  {
	p = p.x*m_tangent + p.y*m_binormal + p.z*m_normal;
  }

  float3 m_tangent;
  float3 m_binormal;
  float3 m_normal;
};


//------------------------------------------------------------------------------
//
// Utility functions 
//
//------------------------------------------------------------------------------


static __forceinline__ __device__ RadiancePRD loadClosesthitRadiancePRD()
{
	RadiancePRD prd = {};

	prd.attenuation.x = __uint_as_float( optixGetPayload_0() );
	prd.attenuation.y = __uint_as_float( optixGetPayload_1() );
	prd.attenuation.z = __uint_as_float( optixGetPayload_2() );
	prd.seed  = optixGetPayload_3();
	return prd;
}

static __forceinline__ __device__ RadiancePRD loadMissRadiancePRD()
{
	RadiancePRD prd = {};
	return prd;
}

static __forceinline__ __device__ void storeClosesthitRadiancePRD( RadiancePRD prd )
{
	optixSetPayload_0(__float_as_uint(prd.attenuation.x));
	optixSetPayload_1(__float_as_uint(prd.attenuation.y));
	optixSetPayload_2(__float_as_uint(prd.attenuation.z));

	optixSetPayload_3(prd.seed);

	optixSetPayload_4(__float_as_uint(prd.emitted.x));
	optixSetPayload_5(__float_as_uint(prd.emitted.y));
	optixSetPayload_6(__float_as_uint(prd.emitted.z));

	optixSetPayload_7(__float_as_uint(prd.distance));

	optixSetPayload_8(__float_as_uint(prd.origin.x));
	optixSetPayload_9(__float_as_uint(prd.origin.y));
	optixSetPayload_10(__float_as_uint(prd.origin.z));

	optixSetPayload_11(__float_as_uint(prd.direction.x));
	optixSetPayload_12(__float_as_uint(prd.direction.y));
	optixSetPayload_13(__float_as_uint(prd.direction.z));

	optixSetPayload_14(prd.done);
}


static __forceinline__ __device__ void storeMissRadiancePRD( RadiancePRD prd )
{
	optixSetPayload_4(__float_as_uint(prd.emitted.x));
	optixSetPayload_5(__float_as_uint(prd.emitted.y));
	optixSetPayload_6(__float_as_uint(prd.emitted.z));

	optixSetPayload_14(prd.done);
}


static __forceinline__ __device__ void cosine_sample_hemisphere(const float u1, const float u2, float3& p)
{
  // Uniformly sample disk.
  const float r   = sqrtf( u1 );
  const float phi = 2.0f*M_PIf * u2;
  p.x = r * cosf( phi );
  p.y = r * sinf( phi );

  // Project up to hemisphere.
  p.z = sqrtf( fmaxf( 0.0f, 1.0f - p.x*p.x - p.y*p.y ) );
}


static __forceinline__ __device__ void traceRadiance(
		OptixTraversableHandle handle,
		float3                 ray_origin,
		float3                 ray_direction,
		float                  tmin,
		float                  tmax,
		RadiancePRD&           prd
		)
{
	unsigned int u0, u1, u2, u3, u4, u5, u6, u7, u8, u9, u10, u11, u12, u13, u14;

	u0 = __float_as_uint( prd.attenuation.x );
	u1 = __float_as_uint( prd.attenuation.y );
	u2 = __float_as_uint( prd.attenuation.z );
	u3 = prd.seed;
	
	// Note:
	// This demonstrates the usage of the OptiX shader execution reordering 
	// (SER) API.  In the case of this computationally simple shading code, 
	// there is no real performance benefit.  However, with more complex shaders
	// the potential performance gains offered by reordering are significant.
	optixTraverse(
		PAYLOAD_TYPE_RADIANCE,
		handle,
		ray_origin,
		ray_direction,
		tmin,
		tmax,
		0.0f,                     // rayTime
		OptixVisibilityMask(1),
		OPTIX_RAY_FLAG_NONE,
		0,                        // SBT offset
		RAY_TYPE_COUNT,           // SBT stride
		0,                        // missSBTIndex
		u0, u1, u2, u3, u4, u5, u6, u7, u8, u9, u10, u11, u12, u13, u14);
	optixReorder(
		// Application specific coherence hints could be passed in here
		);

	optixInvoke(PAYLOAD_TYPE_RADIANCE,
		u0, u1, u2, u3, u4, u5, u6, u7, u8, u9, u10, u11, u12, u13, u14);

	prd.attenuation = make_float3(__uint_as_float(u0), __uint_as_float(u1), __uint_as_float(u2));
	prd.seed = u3;

	prd.emitted = make_float3(__uint_as_float(u4), __uint_as_float(u5), __uint_as_float(u6));
	prd.distance = __uint_as_float(u7);
	prd.origin = make_float3(__uint_as_float(u8), __uint_as_float(u9), __uint_as_float(u10));
	prd.direction = make_float3(__uint_as_float(u11), __uint_as_float(u12), __uint_as_float(u13));
	prd.done = u14;
}


// Returns true if ray is occluded, else false
static __forceinline__ __device__ bool traceOcclusion(
		OptixTraversableHandle handle,
		float3                 ray_origin,
		float3                 ray_direction,
		float                  tmin,
		float                  tmax
		)
{
	// We are only casting probe rays so no shader invocation is needed
	optixTraverse(
		handle,
		ray_origin,
		ray_direction,
		tmin,
		tmax, 0.0f,                // rayTime
		OptixVisibilityMask( 1 ),
		OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT | OPTIX_RAY_FLAG_DISABLE_ANYHIT,
		0,                         // SBT offset
		RAY_TYPE_COUNT,            // SBT stride
		0                          // missSBTIndex
		);
	return optixHitObjectIsHit();
}


//------------------------------------------------------------------------------
//
// Programs
//
//------------------------------------------------------------------------------

extern "C" __global__ void __raygen__rg()
{
	const int    w   = params.width;
	const int    h   = params.height;
	const float3 eye = params.eye;
	const float3 U   = params.U;
	const float3 V   = params.V;
	const float3 W   = params.W;
	const uint3  idx = optixGetLaunchIndex();
	const int    subframe_index = params.subframe_index;

	unsigned int seed = tea<4>( idx.y*w + idx.x, subframe_index );

	float3 result = make_float3(0.0f);
	int i = params.samples_per_launch;

	do
	{
		// The center of each pixel is at fraction (0.5,0.5)
		const float2 subpixel_jitter = make_float2(rnd(seed), rnd(seed));

		const float2 d = 2.0f * make_float2(
			(static_cast<float>(idx.x) + subpixel_jitter.x) / static_cast<float>(w),
			(static_cast<float>(idx.y) + subpixel_jitter.y) / static_cast<float>(h)
		) - 1.0f;
		float3 ray_direction = normalize(d.x * U + d.y * V + W);
		float3 ray_origin    = eye;

		float3 totalAttenuation = make_float3(1.0f);

		RadiancePRD prd;
		prd.distance = 0.0f;
		prd.seed = seed;
		prd.done = false;

		for (int i = 0; i < 8 && !prd.done; i++)
		{
			traceRadiance(
				params.handle,
				ray_origin,
				ray_direction,
				0.01f,  // tmin       // TODO: smarter offset
				1e16f,  // tmax
				prd
			);

			prd.distance = fmaxf(prd.distance, 1.0f);

			result += prd.emitted * totalAttenuation;

			totalAttenuation *= prd.attenuation / (prd.distance * prd.distance);

			ray_origin = prd.origin;
			ray_direction = prd.direction;
		}
	}
	while(--i);

	const uint3    launch_index = optixGetLaunchIndex();
	const unsigned int image_index  = launch_index.y * params.width + launch_index.x;
	float3         accum_color  = result / static_cast<float>( params.samples_per_launch );

	if (subframe_index > 0)
	{
		const float a = 1.0f / static_cast<float>(subframe_index + 1);
		const float3 accum_color_prev = make_float3(params.accum_buffer[image_index]);
		accum_color = lerp(accum_color_prev, accum_color, a);
	}
	params.accum_buffer[image_index] = make_float4(accum_color, 1.0f);
	params.frame_buffer[image_index] = make_color(accum_color);
}


extern "C" __global__ void __miss__radiance()
{
	optixSetPayloadTypes(PAYLOAD_TYPE_RADIANCE);

	MissData* rt_data  = reinterpret_cast<MissData*>(optixGetSbtDataPointer());
	RadiancePRD prd = loadMissRadiancePRD();

	prd.emitted   = make_float3(rt_data->bg_color);
	prd.done      = true;

	storeMissRadiancePRD(prd);
}


extern "C" __global__ void __closesthit__radiance()
{
	optixSetPayloadTypes(PAYLOAD_TYPE_RADIANCE);

	HitGroupData* rt_data = (HitGroupData*)optixGetSbtDataPointer();

	const float2 barycentrics = optixGetTriangleBarycentrics();
	const uint4 tri = params.indices[optixGetPrimitiveIndex()];

	const float3 N_0 = make_float3(params.normals[tri.x]);
	const float3 N_1 = make_float3(params.normals[tri.y]);
	const float3 N_2 = make_float3(params.normals[tri.z]);

	const float3 N =
		N_0 * (1.0f - barycentrics.x - barycentrics.y) +
		N_1 * barycentrics.x +
		N_2 * barycentrics.y;

	const float3 ray_dir = optixGetWorldRayDirection();
	const float3 FFN = faceforward(N, -ray_dir, N);
	const float Tmax = optixGetRayTmax();
	const float3 P = optixGetWorldRayOrigin() + Tmax * ray_dir;

	RadiancePRD prd = loadClosesthitRadiancePRD();

	const float z1 = rnd(prd.seed);
	const float z2 = rnd(prd.seed);

	float3 w_in;
	cosine_sample_hemisphere(z1, z2, w_in);
	Onb onb(FFN);
	onb.inverse_transform(w_in);

	prd.attenuation = rt_data->diffuse_color;
	prd.emitted = rt_data->emission_color;
	prd.distance += fmaxf(fabsf(Tmax) * params.distance_scale, 0.0f);
	prd.origin = P;
	prd.direction = w_in;
	prd.done = false;

	storeClosesthitRadiancePRD(prd);
}
