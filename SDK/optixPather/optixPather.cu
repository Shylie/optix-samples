#include "hip/hip_runtime.h"
//
// Copyright (c) 2023, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
#include <optix.h>

#include "optixPather.h"
#include "random.h"

#include <sutil/vec_math.h>
#include <cuda/helpers.h>

extern "C" {
__constant__ Params params;
}

struct HitInfo
{
	float3 position;
	float3 direction;
	float3 actual_normal;
	float3 normal;
	float distance;
	bool entering;
};

struct BounceResult
{
	float3 attenuation;
	float3 emitted;

	float3 direction;
};

//------------------------------------------------------------------------------
//
// Orthonormal basis helper
//
//------------------------------------------------------------------------------


struct Onb
{
  __forceinline__ __device__ Onb(const float3& normal)
  {
	m_normal = normal;

	if( fabs(m_normal.x) > fabs(m_normal.z) )
	{
	  m_binormal.x = -m_normal.y;
	  m_binormal.y =  m_normal.x;
	  m_binormal.z =  0;
	}
	else
	{
	  m_binormal.x =  0;
	  m_binormal.y = -m_normal.z;
	  m_binormal.z =  m_normal.y;
	}

	m_binormal = normalize(m_binormal);
	m_tangent = cross( m_binormal, m_normal );
  }

  __forceinline__ __device__ void inverse_transform(float3& p) const
  {
	p = p.x*m_tangent + p.y*m_binormal + p.z*m_normal;
  }

  float3 m_tangent;
  float3 m_binormal;
  float3 m_normal;
};


//------------------------------------------------------------------------------
//
// Utility functions 
//
//------------------------------------------------------------------------------


static __forceinline__ __device__ RadiancePRD loadClosesthitRadiancePRD()
{
	RadiancePRD prd = {};

	prd.attenuation.x = __uint_as_float( optixGetPayload_0() );
	prd.attenuation.y = __uint_as_float( optixGetPayload_1() );
	prd.attenuation.z = __uint_as_float( optixGetPayload_2() );
	prd.seed  = optixGetPayload_3();
	return prd;
}

static __forceinline__ __device__ RadiancePRD loadMissRadiancePRD()
{
	RadiancePRD prd = {};
	return prd;
}

static __forceinline__ __device__ void storeClosesthitRadiancePRD( RadiancePRD prd )
{
	optixSetPayload_0(__float_as_uint(prd.attenuation.x));
	optixSetPayload_1(__float_as_uint(prd.attenuation.y));
	optixSetPayload_2(__float_as_uint(prd.attenuation.z));

	optixSetPayload_3(prd.seed);

	optixSetPayload_4(__float_as_uint(prd.emitted.x));
	optixSetPayload_5(__float_as_uint(prd.emitted.y));
	optixSetPayload_6(__float_as_uint(prd.emitted.z));

	optixSetPayload_7(__float_as_uint(prd.distance));

	optixSetPayload_8(__float_as_uint(prd.origin.x));
	optixSetPayload_9(__float_as_uint(prd.origin.y));
	optixSetPayload_10(__float_as_uint(prd.origin.z));

	optixSetPayload_11(__float_as_uint(prd.direction.x));
	optixSetPayload_12(__float_as_uint(prd.direction.y));
	optixSetPayload_13(__float_as_uint(prd.direction.z));

	optixSetPayload_14(prd.done);
}


static __forceinline__ __device__ void storeMissRadiancePRD( RadiancePRD prd )
{
	optixSetPayload_4(__float_as_uint(prd.emitted.x));
	optixSetPayload_5(__float_as_uint(prd.emitted.y));
	optixSetPayload_6(__float_as_uint(prd.emitted.z));

	optixSetPayload_14(prd.done);
}


static __forceinline__ __device__ void cosine_sample_hemisphere(const float u1, const float u2, float3& p)
{
  // Uniformly sample disk.
  const float r   = sqrtf( u1 );
  const float phi = 2.0f*M_PIf * u2;
  p.x = r * cosf( phi );
  p.y = r * sinf( phi );

  // Project up to hemisphere.
  p.z = sqrtf( fmaxf( 0.0f, 1.0f - p.x*p.x - p.y*p.y ) );
}


static __forceinline__ __device__ void traceRadiance(
		OptixTraversableHandle handle,
		float3                 ray_origin,
		float3                 ray_direction,
		float                  tmin,
		float                  tmax,
		RadiancePRD&           prd
		)
{
	unsigned int u0, u1, u2, u3, u4, u5, u6, u7, u8, u9, u10, u11, u12, u13, u14;

	u0 = __float_as_uint(prd.attenuation.x);
	u1 = __float_as_uint(prd.attenuation.y);
	u2 = __float_as_uint(prd.attenuation.z);
	u3 = prd.seed;
	
	// Note:
	// This demonstrates the usage of the OptiX shader execution reordering 
	// (SER) API.  In the case of this computationally simple shading code, 
	// there is no real performance benefit.  However, with more complex shaders
	// the potential performance gains offered by reordering are significant.
	optixTraverse(
		PAYLOAD_TYPE_RADIANCE,
		handle,
		ray_origin,
		ray_direction,
		tmin,
		tmax,
		0.0f,                     // rayTime
		OptixVisibilityMask(1),
		OPTIX_RAY_FLAG_NONE,
		0,                        // SBT offset
		RAY_TYPE_COUNT,           // SBT stride
		0,                        // missSBTIndex
		u0, u1, u2, u3, u4, u5, u6, u7, u8, u9, u10, u11, u12, u13, u14);

	//optixReorder(reinterpret_cast<CommonData*>(optixHitObjectGetSbtDataPointer())->material_type, 2);
	//optixReorder();

	optixInvoke(PAYLOAD_TYPE_RADIANCE,
		u0, u1, u2, u3, u4, u5, u6, u7, u8, u9, u10, u11, u12, u13, u14);

	prd.attenuation = make_float3(__uint_as_float(u0), __uint_as_float(u1), __uint_as_float(u2));
	prd.seed = u3;

	prd.emitted = make_float3(__uint_as_float(u4), __uint_as_float(u5), __uint_as_float(u6));
	prd.distance = __uint_as_float(u7);
	prd.origin = make_float3(__uint_as_float(u8), __uint_as_float(u9), __uint_as_float(u10));
	prd.direction = make_float3(__uint_as_float(u11), __uint_as_float(u12), __uint_as_float(u13));
	prd.done = u14;
}

//------------------------------------------------------------------------------
//
// Programs
//
//------------------------------------------------------------------------------

extern "C" __global__ void __raygen__rg()
{
	const int    w   = params.width;
	const int    h   = params.height;
	const float3 eye = params.eye;
	const float3 U   = params.U;
	const float3 V   = params.V;
	const float3 W   = params.W;
	const uint3  idx = optixGetLaunchIndex();
	const int    subframe_index = params.subframe_index;

	unsigned int seed = tea<4>(idx.y * w + idx.x, subframe_index);

	float3 result = make_float3(0.0f);
	int i = params.samples_per_launch;

	do
	{
		// The center of each pixel is at fraction (0.5,0.5)
		const float2 subpixel_jitter = make_float2(rnd(seed), rnd(seed));

		const float2 d = 2.0f * make_float2(
			(static_cast<float>(idx.x) + subpixel_jitter.x) / static_cast<float>(w),
			(static_cast<float>(idx.y) + subpixel_jitter.y) / static_cast<float>(h)
		) - 1.0f;
		float3 ray_direction = normalize(d.x * U + d.y * V + W);
		float3 ray_origin    = eye;

		float3 total_attenuation = make_float3(1.0f);

		RadiancePRD prd;
		prd.distance = 0.0f;
		prd.seed = seed;
		prd.done = false;

		for (int j = 0; j < 8 && !prd.done; j++)
		{
			traceRadiance(
				params.handle,
				ray_origin,
				ray_direction,
				0.01f,  // tmin       // TODO: smarter offset
				1e16f,  // tmax
				prd
			);

			result += prd.emitted * total_attenuation;

			const float clamped_distance = fmaxf(prd.distance, 1.0f);

			total_attenuation *= prd.attenuation / (clamped_distance * clamped_distance);

			ray_origin = prd.origin;
			ray_direction = prd.direction;
		}
	}
	while(--i);

	const uint3    launch_index = optixGetLaunchIndex();
	const unsigned int image_index  = launch_index.y * params.width + launch_index.x;
	float3         accum_color  = result / static_cast<float>(params.samples_per_launch);

	if (subframe_index > 0)
	{
		const float a = 1.0f / static_cast<float>(subframe_index + 1);
		const float3 accum_color_prev = make_float3(params.accum_buffer[image_index]);
		accum_color = lerp(accum_color_prev, accum_color, a);
	}

	params.accum_buffer[image_index] = make_float4(accum_color, 1.0f);
	params.frame_buffer[image_index] = make_float4(accum_color, 1.0f);
}


extern "C" __global__ void __miss__radiance()
{
	optixSetPayloadTypes(PAYLOAD_TYPE_RADIANCE);

	MissData* rt_data  = reinterpret_cast<MissData*>(optixGetSbtDataPointer());
	RadiancePRD prd = loadMissRadiancePRD();

	prd.emitted   = make_float3(rt_data->bg_color);
	prd.done      = true;

	storeMissRadiancePRD(prd);
}


extern "C" __global__ void __closesthit__radiance()
{
	optixSetPayloadTypes(PAYLOAD_TYPE_RADIANCE);

	HitGroupData* rt_data = (HitGroupData*)optixGetSbtDataPointer();

	const float2 barycentrics = optixGetTriangleBarycentrics();
	const uint4 tri = params.indices[optixGetPrimitiveIndex()];

	const float3 N_0 = make_float3(params.normals[tri.x]);
	const float3 N_1 = make_float3(params.normals[tri.y]);
	const float3 N_2 = make_float3(params.normals[tri.z]);

	const float3 actual_normal =
		N_0 * (1.0f - barycentrics.x - barycentrics.y) +
		N_1 * barycentrics.x +
		N_2 * barycentrics.y;

	const float3 ray_dir = optixGetWorldRayDirection();
	const float sign = copysignf(1.0f, dot(-ray_dir, actual_normal));
	const float3 forward_facing_normal = actual_normal * sign;
	const float t_max = optixGetRayTmax();
	const float3 P = optixGetWorldRayOrigin() + t_max * ray_dir;

	RadiancePRD prd = loadClosesthitRadiancePRD();

	const float scaled_distance = fabsf(t_max) * params.distance_scale;

	HitInfo hit;
	hit.position = P;
	hit.direction = ray_dir;
	hit.actual_normal = actual_normal;
	hit.normal = forward_facing_normal;
	hit.distance = fmaxf(scaled_distance, 1.0f);
	hit.entering = sign > 0.0f;

	BounceResult br = optixDirectCall<BounceResult, const HitInfo&, const HitGroupData&, unsigned int&>(
		rt_data->common.material_type, hit, *rt_data, prd.seed
	);

	prd.attenuation = br.attenuation;
	prd.emitted = br.emitted;
	prd.distance += scaled_distance;
	prd.origin = P;
	prd.direction = br.direction;
	prd.done = false;

	storeClosesthitRadiancePRD(prd);
}

extern "C" __device__ BounceResult __direct_callable__lambertian(const HitInfo& hit, const HitGroupData& material, unsigned int& seed)
{
	const float z1 = rnd(seed);
	const float z2 = rnd(seed);

	float3 w_in;
	cosine_sample_hemisphere(z1, z2, w_in);
	const Onb onb(hit.normal);
	onb.inverse_transform(w_in);

	return
	{
		material.lambertian.attenuation,
		material.lambertian.emitted,
		w_in
	};
}

extern "C" __device__ BounceResult __direct_callable__metal(const HitInfo& hit, const HitGroupData& material, unsigned int& seed)
{
	return
	{
		material.metal.attenuation,
		material.metal.emitted,
		reflect(hit.direction, hit.normal)
	};
}

extern "C" __device__ BounceResult __direct_callable__glass(const HitInfo& hit, const HitGroupData& material, unsigned int& seed)
{
	float3 new_direction;
	if (!refract(new_direction, hit.direction, hit.normal, material.glass.refractive_index))
	{
		new_direction = reflect(hit.direction, hit.normal);
	}
	else
	{
		float cos_theta = dot(hit.direction, hit.normal);
		if (cos_theta < 0.0f)
		{
			cos_theta = -cos_theta;
		}
		else
		{
			cos_theta = dot(new_direction, hit.normal);
		}

		if (rnd(seed) <= fresnel_schlick(cos_theta, material.glass.refractive_index))
		{
			new_direction = reflect(hit.direction, hit.normal);
		}
	}

	float3 attenuation;
	if (hit.entering)
	{
		attenuation = material.glass.attenuation;
	}
	else
	{
		const float3 absorbance = (1.0f - material.glass.attenuation) * hit.distance;
		attenuation = expf(-absorbance);
	}

	return
	{
		attenuation,
		material.glass.emitted,
		new_direction
	};
}

extern "C" __device__ BounceResult __direct_callable__test(const HitInfo & hit, const HitGroupData & material, unsigned int& seed)
{
	const float multiplier = clamp(dot(hit.normal, -hit.direction), 0.0f, 1.0f);

	float3 w_in;
	cosine_sample_hemisphere(multiplier, 1.0f - multiplier, w_in);
	const Onb onb(hit.normal);
	onb.inverse_transform(w_in);

	const Onb onb2(w_in);
	cosine_sample_hemisphere(1.0f - multiplier, multiplier, w_in);
	onb.inverse_transform(w_in);

	return
	{
		material.test.attenuation,
		material.test.emitted,
		reflect(hit.direction, w_in)
	};
}